#include "hip/hip_runtime.h"
#include "Solver.cuh"

#include <iterator>
#include <algorithm>
#include <set>

#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>

#include <thrust/binary_search.h>
#include <thrust/sort.h>

#include <thrust/copy.h>
#include <thrust/transform.h>
#include <thrust/transform.h>

#include <thrust/functional.h>

#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/counting_iterator.h>

// x, y, i default to -1
__host__ __device__
Particle::Particle( float x, float y, float i, int idx ) :
z( idx ),
peakIntensity( i ),
nContributingParticles( 1 ),
lastContributingsliceIdx( idx ),
pState( Particle::State::NO_MATCH )
{
	this->x = x;
	this->y = y;
	this->i = i;
}

Solver::Solver() :
m_uMaskRadius( 0 ),
m_uFeatureRadius( 0 ),
m_uMaxStackCount( 0 ),
m_uNeighborRadius( 0 ),
m_uCurPrevParticleCount( 0 )
{
}

Solver::Solver( uint32_t mR, uint32_t fR, uint32_t minSC, uint32_t maxSC, uint32_t nR ) :
m_uMaskRadius( mR ),
m_nMaxLevel( 3 ),
m_uFeatureRadius( fR ),
m_uMinStackCount( minSC ),
m_uMaxStackCount( maxSC ),
m_uNeighborRadius( nR ),
m_uCurPrevParticleCount( 0 )
{
	// Neighbor region diameter
	int diameter = 2 * m_uMaskRadius + 1;

	// Make host mats
	cv::Mat h_Circ( cv::Size( diameter, diameter ), CV_32F, 0.f );
	cv::Mat h_RX( cv::Size( diameter, diameter ), CV_32F, 0.f );
	cv::Mat h_RY( cv::Size( diameter, diameter ), CV_32F, 0.f );
	cv::Mat h_R2( cv::Size( diameter, diameter ), CV_32F, 0.f );

	// set up circle mask
	cv::circle( h_Circ, cv::Point( m_uMaskRadius, m_uMaskRadius ), m_uMaskRadius, 1.f, -1 );

	// set up Rx and part of r2
	for ( int y = 0; y < diameter; y++ )
	{
		for ( int x = 0; x < diameter; x++ )
		{
			cv::Point p( x, y );
			h_RX.at<float>( p ) = x + 1;
			h_RY.at<float>( p ) = y + 1;
			h_R2.at<float>( p ) = pow( -(float) m_uMaskRadius + x, 2 ) + pow( -(float) m_uMaskRadius + y, 2 );
		}
	}

	// I forget what these do...
	cv::threshold( h_R2, h_R2, pow( (double) m_uMaskRadius, 2 ), 1, cv::THRESH_TOZERO_INV );
	cv::multiply( h_RX, h_Circ, h_RX );
	cv::multiply( h_RY, h_Circ, h_RY );

	// Upload to gpu mats
	//h_Circ.copyTo( m_dCircleMask );
	//h_RX.copyTo( m_dRadXKernel );
	//h_RY.copyTo( m_dRadYKernel );
	//h_R2.copyTo( m_dRadSqKernel );

	m_dCircleMask.upload( h_Circ );
	m_dRadXKernel.upload( h_RX );
	m_dRadYKernel.upload( h_RY );
	m_dRadSqKernel.upload( h_R2 );
}

__host__ __device__
int pixelToGridIdx( float x, float y, int N, int m )
{
	const int cellSize = N >> m;
	const int cellCount = N / cellSize;

	int cellX = x / cellSize;
	int cellY = y / cellSize;

	int cellIdx = cellX + cellCount * cellY;
	return cellIdx;
}

__host__ __device__
int pixelToGridIdx( Particle p, int N, int m )
{
	return pixelToGridIdx( p.x, p.y, N, m );
}

struct PixelToGridIdx : public thrust::unary_function < Particle, int >
{
	int N; // Image size
	int M; // division level

	PixelToGridIdx( int n, int m ) :N( n ), M( m ) {}

	__host__ __device__
	int operator()( const Particle& p )
	{
		return pixelToGridIdx( p.x, p.y, N, M );
	}
};

struct IsParticleAtIdx
{
	int N;
	int featureRad;
	IsParticleAtIdx( int n, int k ) : N( n ), featureRad( k ) {}

	template <typename tuple_t>
	__host__ __device__
	bool operator()( tuple_t T )
	{
		unsigned char val = thrust::get<0>( T );
		int idx = thrust::get<1>( T );
		int x = idx % N;
		int y = idx / N;
		
		// We care if the pixel is nonzero and its within the kernel radius
		return ( val != 0 ) && ( x > featureRad ) && ( y > featureRad ) && ( x + featureRad < N ) && ( y + featureRad < N );
	}
};

struct MakeParticleFromIdx
{
	int sliceIdx;
	int kernelRad;
	int N;

	float * lmImg;
	float * circKernel;
	float * rxKernel;
	float * ryKernel;
	float * rSqKernel;

	MakeParticleFromIdx( int sIdx, int n, int kRad, float * lm, float * cK, float * xK, float * yK, float * sqK ) :
		sliceIdx( sIdx ),
		N(n),
		kernelRad(kRad),
		lmImg( lm ),
		circKernel( cK ),
		rxKernel( xK ),
		ryKernel( yK ),
		rSqKernel( sqK )
	{
	}

	__host__ __device__
	Particle operator()( int idx )
	{
		// Grab x, y values
		int x = idx % N;
		int y = idx / N;

		// Make tmp pointers to our kernels and advance them as we iterate
		float * tmpCircKernPtr = circKernel;
		float * tmpXKernPtr = rxKernel;
		float * tmpYKernPtr = ryKernel;

		// To be calculated
		float total_mass( 0 );
		float x_offset( 0 ), y_offset( 0 );

		// Perform the multiplcations
		for ( int iY = -kernelRad; iY <= kernelRad; iY++ )
		{
			// For y, go down then up
			float * ptrY = &lmImg[idx + ( N * iY )];
			for ( int iX = -kernelRad; iX <= kernelRad; iX++ )
			{
				// Get the local max img value
				float lmImgVal = ptrY[iX]; 

				// Multiply by kernel, sum, advance kernel pointer
				total_mass += lmImgVal * ( *tmpCircKernPtr++ );
				x_offset += lmImgVal * ( *tmpXKernPtr++ );
				y_offset += lmImgVal * ( *tmpYKernPtr++ );
			}
		}

		// Calculate x val, y val
		float x_val = float(x) + x_offset / total_mass;
		float y_val = float(y) + y_offset / total_mass;

		// Construct particle and return
		Particle p( x_val, y_val, total_mass, sliceIdx );
		return p;
	}
};

struct ParticleMatcher
{
	int N;
	int M;
	int sliceIdx;
	int maxStackCount;
	float neighborRadius;

	int * cellLowerBound;
	int * cellUpperBound;

	Particle* prevParticles;

	ParticleMatcher( int n, int m, int s, int mSC, int nR, int * cLB, int * cUB, Particle * pP ) :
		N( n ),
		M( m ),
		sliceIdx( s ),
		maxStackCount( mSC ),
		neighborRadius( nR ),
		cellLowerBound( cLB ),
		cellUpperBound( cUB ),
		prevParticles( pP )
	{
	}

	// Returns null if no match is found
	__host__ __device__
	Particle * operator()( Particle newParticle )
	{
		// There are a total of 9 cells we might have to search. last is sentinel
		int cellIndices[10] = { -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 };

		// But we always search at least one
		cellIndices[0] = pixelToGridIdx( newParticle, N, M );

		// Neighbors to follow
		Particle * pBestMatch = nullptr;
		for ( int c = 0; cellIndices[c] >= 0; c++ )
		{
			// It would be nice to parallelize around this, but probably not worth it
			int cellIdx = cellIndices[c];
			int lower = cellLowerBound[cellIdx];
			int upper = cellUpperBound[cellIdx];
			for ( int p = lower; p < upper; p++ )
			{
				Particle& oldParticle = prevParticles[p];

				// tooFar might not be necessary if I cull beforehand
				bool tooFar = ( sliceIdx - oldParticle.lastContributingsliceIdx != 1 );
				bool tooMany = ( oldParticle.nContributingParticles > maxStackCount );
				bool alreadyDone = ( oldParticle.pState == Particle::State::SEVER );
				if ( tooFar || tooMany || alreadyDone )
					continue;

				// See if the particle is within our range
				float dX = oldParticle.x - newParticle.x;
				float dY = oldParticle.y - newParticle.y;
				float distSq = pow( dX, 2 ) + pow( dY, 2 );

				if ( distSq < neighborRadius * neighborRadius )
				{
					// If there already was a match, see if this one is better
					if ( pBestMatch )
					{
						// Find the old distance
						dX = pBestMatch->x - newParticle.x;
						dY = pBestMatch->y - newParticle.y;

						// If this one is closer, assign it as the match
						if ( pow( dX, 2 ) + pow( dY, 2 ) > distSq )
							pBestMatch = &oldParticle;
					}
					else 
						pBestMatch = &oldParticle;
				}
			}
		}

		// Could check sever state here

		return pBestMatch;
	}
};

struct CheckIfMatchIsNotNull
{
	template <typename tuple_t>
	__host__ __device__
	bool operator()( const tuple_t T )
	{
		Particle * pMatch = thrust::get<1>( T );
		return pMatch != nullptr;
	}
};
struct CheckIfMatchIsNull
{
	template <typename tuple_t>
	__host__ __device__
		bool operator()( const tuple_t T )
	{
		Particle * pMatch = thrust::get<1>( T );
		return pMatch == nullptr;
	}
};

// This gets called on matched particles and handles intensity state logic
// You should ensure this is thread safe beforehand, somehow (remove duplicates? not really sure)
struct UpdateMatchedParticle
{
	int sliceIdx;

	UpdateMatchedParticle( int s ) : sliceIdx( s ) {}

	// This kind of thing could be parallelized in a smarter way, probably
	template <typename tuple_t>
	__host__ __device__
	int operator()( const tuple_t T )
	{
		Particle newParticle = thrust::get<0>( T );
		Particle * oldParticle = thrust::get<1>( T );
		switch ( oldParticle->pState )
		{
			case Particle::State::NO_MATCH:
				// Shouldn't ever get no match, but assign the state and fall through
				oldParticle->pState = Particle::State::INCREASING;
			case Particle::State::INCREASING:
				// If we're increasing, see if the new guy prompts a decrease
				// Should we check to see if more than one particle has contributed?
				if ( oldParticle->i > newParticle.i )
					oldParticle->pState = Particle::State::DECREASING;
				// Otherwise see if we should update the peak intensity and z position
				else if ( newParticle.i > oldParticle->peakIntensity )
				{
					oldParticle->peakIntensity = newParticle.i;
					oldParticle->z = (float) sliceIdx;
				}
				break;
			case Particle::State::DECREASING:
				// In this case, if it's still decreasing then fall through
				if ( oldParticle->i > newParticle.i )
					break;
				// If we're severing, assing the state and fall through
				oldParticle->pState = Particle::State::SEVER;

				// I could probably catch this earlier
			case Particle::State::SEVER:
				// Continue here (could catch this earlier)
				oldParticle = nullptr;
		}

		// could do this in yet another call, if you were so inclined
		// If we didn't sever and null out above
		if ( oldParticle != nullptr )
		{
			// It's a match, bump the particle count and compute an averaged position (?)
			oldParticle->nContributingParticles++;
			oldParticle->lastContributingsliceIdx = sliceIdx;

			// I don't know about the averaged position thing
			oldParticle->x = 0.5f * ( oldParticle->x + newParticle.x );
			oldParticle->y = 0.5f * ( oldParticle->y + newParticle.y );
		}

		return 0;
	}
};

struct IsParticleUnmatched
{
	__host__ __device__
	bool operator()( const Particle p )
	{
		return p.pState == Particle::State::NO_MATCH;
	}
};

struct ParticleOrderingComp
{
	int N, M;
	ParticleOrderingComp( int n, int m ) : N( n ), M( m ) {}

	__host__ __device__
	bool operator()( const Particle a, const Particle b )
	{
		return pixelToGridIdx( a, N, M ) < pixelToGridIdx( b, N, M );
	}
};

struct MaybeRemoveParticle
{
	int sliceIdx;
	int minSlices;
	MaybeRemoveParticle( int s, int m ) : sliceIdx( s ), minSlices( m ) {}

	__host__ __device__
	bool operator()(const Particle p)
	{
		return ( sliceIdx - p.lastContributingsliceIdx > 2 && ( p.pState != Particle::State::SEVER || p.nContributingParticles < minSlices ) );
	}
};

struct IsNonzero
{
	bool operator()( const unsigned char val )
	{
		return val != 0;
	}
};

uint32_t Solver::FindParticles( Datum& D )
{
	const int N = D.d_LocalMaxImg.rows;
	const int m = m_nMaxLevel;
	const int cellSize = N >> m;
	const int cellCount = N / cellSize;
	const int nTotalCells = cellCount * cellCount;

	dFloatptr d_pLocalMaxImgBuf( D.d_LocalMaxImg.ptr<float>() );
	dFloatptr d_pCirleKernel( m_dCircleMask.ptr<float>() );
	dFloatptr d_pRxKernel( m_dRadXKernel.ptr<float>() );
	dFloatptr d_pRyKernel( m_dRadYKernel.ptr<float>() );
	dFloatptr d_pR2Kernel( m_dRadSqKernel.ptr<float>() );

	// Cull the herd
	int minSlices = 3;
	auto itLastPrevParticleEnd = md_PrevParticleVec.begin() + m_uCurPrevParticleCount;
	itLastPrevParticleEnd = thrust::remove_if( md_PrevParticleVec.begin(), itLastPrevParticleEnd, MaybeRemoveParticle( D.sliceIdx, minSlices ) );
	m_uCurPrevParticleCount = itLastPrevParticleEnd - md_PrevParticleVec.begin();

	// Make a device vector out of the particle buffer pointer (it's contiguous)
	dUcharPtr d_pParticleImgBuf( D.d_ParticleImg.ptr<unsigned char>() );
	dUcharVec d_ParticleImgVec( d_pParticleImgBuf, d_pParticleImgBuf + D.d_ParticleImg.size().area() );

	// For each pixel in the particle image, we care if it's nonzero and if it's far enough from the edges
	// So we need its index (transformable into twoD pos) and its value
	auto itDetectParticleBegin = thrust::make_zip_iterator( thrust::make_tuple( d_ParticleImgVec.begin(), thrust::counting_iterator<int>( 0 ) ) );
	auto itDetectParticleEnd = thrust::make_zip_iterator( thrust::make_tuple( d_ParticleImgVec.end(), thrust::counting_iterator<int>( N*N ) ) );

	// Then, if the particle fits our criteria, we copy its index (from the counting iterator) into this vector, and discard the uchar
	dIntVec d_NewParticleIndicesVec( N*N );
	auto itFirstNewParticle = thrust::make_zip_iterator( thrust::make_tuple( thrust::discard_iterator<>(), d_NewParticleIndicesVec.begin() ) );
	auto itLastNewParticle = thrust::copy_if( itDetectParticleBegin, itDetectParticleEnd, itFirstNewParticle, IsParticleAtIdx( N, m_uFeatureRadius ) );
	size_t newParticleCount = itLastNewParticle - itFirstNewParticle;

	// Now transform each index into a particle by looking at values inside the lmimg and using the kernels
	dParticleVec d_NewParticleVec( newParticleCount );
	thrust::transform( d_NewParticleIndicesVec.begin(), d_NewParticleIndicesVec.begin() + newParticleCount, d_NewParticleVec.begin(),
					   MakeParticleFromIdx( D.sliceIdx, N, m_uMaskRadius, d_pLocalMaxImgBuf.get(), d_pCirleKernel.get(), d_pRxKernel.get(), d_pRyKernel.get(), d_pR2Kernel.get() ) );

	// The grid cell vec might be split into two vecs like this (they should also be class members, but I'll do that later)
	dIntVec d_GridCellLowerBoundsVec( nTotalCells ), d_GridCellUpperBoundsVec( nTotalCells );

	// Initialize grid cells
	using particleIter = dParticleVec::iterator;
	using pixelToGridIdxIter = thrust::transform_iterator < PixelToGridIdx, particleIter >;

	pixelToGridIdxIter itPrevParticleBegin = thrust::make_transform_iterator<PixelToGridIdx, particleIter>( md_PrevParticleVec.begin(), PixelToGridIdx( N, m ) );
	pixelToGridIdxIter itPrevParticleEnd = thrust::make_transform_iterator<PixelToGridIdx, particleIter>( itLastPrevParticleEnd, PixelToGridIdx( N, m ) );

	thrust::lower_bound( itPrevParticleBegin, itPrevParticleEnd, thrust::counting_iterator<int>( 0 ), thrust::counting_iterator<int>( nTotalCells ), d_GridCellLowerBoundsVec.begin() );
	thrust::upper_bound( itPrevParticleBegin, itPrevParticleEnd, thrust::counting_iterator<int>( 0 ), thrust::counting_iterator<int>( nTotalCells ), d_GridCellUpperBoundsVec.begin() );

	// Tranform new particles into a vector of particle pointers; if they are null then no match was found (?)
	dParticlePtrVec d_ParticleMatchVec( newParticleCount );
	thrust::transform( d_NewParticleVec.begin(), d_NewParticleVec.end(), d_ParticleMatchVec.begin(),
					   ParticleMatcher( N, m, D.sliceIdx, m_uMaxStackCount, m_uNeighborRadius, d_GridCellLowerBoundsVec.data().get(), d_GridCellUpperBoundsVec.data().get(), md_PrevParticleVec.data().get() ) );

	// Zip the pointer vec and newparticle vec
	auto itNewParticleToMatchedParticleBegin = thrust::make_zip_iterator( thrust::make_tuple( d_NewParticleVec.begin(), d_ParticleMatchVec.begin() ) );
	auto itNewParticleToMatchedParticleEnd = thrust::make_zip_iterator( thrust::make_tuple( d_NewParticleVec.end(), d_ParticleMatchVec.end() ) );

	// If there was a match, update the intensity state. I don't know how to do a for_each_if other than a transform_if that discards the output
	thrust::transform_if( itNewParticleToMatchedParticleBegin, itNewParticleToMatchedParticleEnd, thrust::discard_iterator<>(), UpdateMatchedParticle( D.sliceIdx ), CheckIfMatchIsNotNull() );

	// Copy all unmatched particles into a new vector; we copy a tuple of new particles and pointers to matches, discarding the pointers
	dParticleVec d_UnmatchedParticleVec( newParticleCount );
	auto itNewParticleAndPrevParticleMatchBegin = thrust::make_zip_iterator( thrust::make_tuple( d_UnmatchedParticleVec.begin(), thrust::discard_iterator<>() ) );

	// Copy new particles if their corresponding match is null
	auto itNewParticleAndPrevParticleMatchEnd = thrust::copy_if( itNewParticleToMatchedParticleBegin, itNewParticleToMatchedParticleEnd, itNewParticleAndPrevParticleMatchBegin, CheckIfMatchIsNull() );
	size_t numUnmatchedParticles = itNewParticleAndPrevParticleMatchEnd - itNewParticleAndPrevParticleMatchBegin;

	// We have two options here; the easy option is to just tack these new particles onto the previous particle vector and sort the whole thing
	// alternatively you could set a flag in previous particles if the matching process caused them to move to a new grid cell and then treat those particles as unmatched
	// you could then sort the unmatched particles (relatively few compared to the count of previous particles) and then merge them into the prev particle vec, which is still sorted

	// Option A
	{
		// first make room for the new particles, if we need it
		size_t newPrevParticleCount = numUnmatchedParticles + m_uCurPrevParticleCount;
		if ( newPrevParticleCount > md_PrevParticleVec.size() )
			md_PrevParticleVec.resize( newPrevParticleCount );

		// copy unmatched particles onto the original end of the previous particle vec
		auto itNewParticleDest = md_PrevParticleVec.begin() + m_uCurPrevParticleCount;
		thrust::copy( d_UnmatchedParticleVec.begin(), d_UnmatchedParticleVec.end(), itNewParticleDest );

		// Sort the whole thing
		thrust::sort( md_PrevParticleVec.begin(), md_PrevParticleVec.begin() + newPrevParticleCount, ParticleOrderingComp( N, m ) );
		m_uCurPrevParticleCount = newPrevParticleCount;
	}

	// Option B; NYI

	std::cout << "Slice Idx:\t" << D.sliceIdx << "\tNew Particles:\t" << newParticleCount << "\tUnmatched Particles:\t" << numUnmatchedParticles << "\tFound Particles:\t" << m_uCurPrevParticleCount << std::endl;

	return m_uCurPrevParticleCount;
}

std::vector<Particle> Solver::GetFoundParticles() const
{

	//int nParticles = std::count_if( m_vPrevParticles.begin(), m_vPrevParticles.end(), [] ( const Particle& p ) {
	//	return p.pState == Particle::State::SEVER && p.nContributingParticles > 2;
	//} );
	//std::cout << "Final particle count: " << nParticles << std::endl;

	std::vector<Particle> ret;
	//( m_vFoundParticles.size() );
	//std::transform( m_vFoundParticles.begin(), m_vFoundParticles.end(), ret.begin(),
	//				[] ( const ParticleStack& pS ) {return pS.GetRefinedParticle(); } );
	return ret;
}