#include "hip/hip_runtime.h"
#include "CenterFind.h"
#include <iterator>
#include <algorithm>
#include <set>

#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>

#include <thrust/binary_search.h>
#include <thrust/sort.h>

#include <thrust/copy.h>
#include <thrust/transform.h>
#include <thrust/transform.h>

#include <thrust/functional.h>

#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/counting_iterator.h>

// x, y, i default to -1
__host__ __device__
Particle::Particle( float x, float y, float i, int idx ) :
z( idx ),
peakIntensity( i ),
nContributingParticles( 1 ),
lastContributingsliceIdx( idx ),
pState( Particle::State::NO_MATCH )
{
	this->x = x;
	this->y = y;
	this->i = i;
}

Solver::Solver() :
m_uMaskRadius( 0 ),
m_uFeatureRadius( 0 ),
m_uMaxStackCount( 0 ),
m_uNeighborRadius( 0 )
{
}

Solver::Solver( uint32_t mR, uint32_t fR, uint32_t minSC, uint32_t maxSC, uint32_t nR ) :
m_uMaskRadius( mR ),
m_nMaxLevel( 3 ),
m_uFeatureRadius( fR ),
m_uMinStackCount( minSC ),
m_uMaxStackCount( maxSC ),
m_uNeighborRadius( nR )
{
	// Neighbor region diameter
	int diameter = 2 * m_uMaskRadius + 1;

	// Make host mats
	cv::Mat h_Circ( cv::Size( diameter, diameter ), CV_32F, 0.f );
	cv::Mat h_RX = h_Circ;
	cv::Mat h_RY = h_Circ;
	cv::Mat h_R2 = h_Circ;

	// set up circle mask
	cv::circle( h_Circ, cv::Point( m_uMaskRadius, m_uMaskRadius ), m_uMaskRadius, 1.f, -1 );

	// set up Rx and part of r2
	for ( int i = 0; i < diameter; i++ )
	{
		for ( int j = 0; j < diameter; j++ )
		{
			h_RX.at<float>( i, j ) = float( j + 1 );
			h_R2.at<float>( i, j ) += float( pow( j - m_uMaskRadius, 2 ) );
		}
	}

	// set up Ry and the rest of r2
	for ( int i = 0; i < diameter; i++ )
	{
		for ( int j = 0; j < diameter; j++ )
		{
			h_RY.at<float>( i, j ) = float( i + 1 );
			h_R2.at<float>( i, j ) += float( pow( i - m_uMaskRadius, 2 ) );
		}
	}

	// I forget what these do...
	cv::threshold( h_R2, h_R2, pow( (double) m_uMaskRadius, 2 ), 1, cv::THRESH_TOZERO_INV );
	cv::multiply( h_RX, h_Circ, h_RX );
	cv::multiply( h_RY, h_Circ, h_RY );

	// Upload to gpu mats
	m_dCircleMask.upload( h_Circ );
	m_dRadXKernel.upload( h_RX );
	m_dRadYKernel.upload( h_RY );
	m_dRadSqKernel.upload( h_R2 );
}

__host__ __device__
int pixelToGridIdx( float x, float y, int N, int m )
{
	const int cellSize = N >> m;
	const int cellCount = N / cellSize;

	int cellX = x / cellSize;
	int cellY = y / cellSize;

	int cellIdx = cellX + cellCount * cellY;
	return cellIdx;
}

__host__ __device__
int pixelToGridIdx( Particle p, int N, int m )
{
	return pixelToGridIdx( p.x, p.y, N, m );
}

struct PixelToGridIdx : public thrust::unary_function < Particle, int >
{
	int N; // Image size
	int M; // division level

	PixelToGridIdx( int n, int m ) :N( n ), M( m ) {}

	__host__ __device__
	int operator()( const Particle& p )
	{
		return pixelToGridIdx( p.x, p.y, N, M );
	}
};

struct IsParticleAtIdx
{
	int N;
	int kernelRad;
	IsParticleAtIdx( int n, int k ) : N( n ), kernelRad(k) {}

	template <typename tuple_t>
	__host__ __device__
	bool operator()( tuple_t T )
	{
		unsigned char val = thrust::get<0>( T );
		int idx = thrust::get<1>( T );
		int x = idx % N;
		int y = idx / N;
		
		// We care if the pixel is nonzero and its within the kernel radius
		return ( val != 0 ) && ( x > kernelRad ) && ( y > kernelRad ) && ( x + kernelRad < N ) && ( y + kernelRad < N );
	}
};

struct MakeParticleFromIdx
{
	int sliceIdx;
	int kernelRad;
	int N;

	float * lmImg;
	float * circKernel;
	float * rxKernel;
	float * ryKernel;
	float * rSqKernel;

	MakeParticleFromIdx( int sIdx, int n, int kRad, float * lm, float * cK, float * xK, float * yK, float * sqK ) :
		sliceIdx( sIdx ),
		N(n),
		kernelRad(kRad),
		lmImg( lm ),
		circKernel( cK ),
		rxKernel( xK ),
		ryKernel( yK ),
		rSqKernel( sqK )
	{
	}

	template <typename tuple_t>
	__host__ __device__
	Particle operator()( tuple_t T )
	{
		unsigned char val = thrust::get<0>( T );
		int idx = thrust::get<1>( T );
		int x = idx % N;
		int y = idx / N;

		float total_mass( 0 );
		float x_offset( 0 ), y_offset( 0 );

		float * tmpCircKernPtr = circKernel;
		float * tmpXKernPtr = rxKernel;
		float * tmpYKernPtr = ryKernel;

		for ( int iY = -kernelRad; iY <= kernelRad; iY++ )
		{
			// For y, go down then up
			float * ptrY = &lmImg[idx - ( N * iY )];
			for ( int iX = -kernelRad; iX <= kernelRad; iX++ )
			{
				// Get the local max img value
				float lmImgVal = ptrY[iX]; 

				// Multiply by kernel, sum, advance kernel pointer
				total_mass += lmImgVal * ( *tmpCircKernPtr++ );
				x_offset += lmImgVal * ( *tmpXKernPtr++ );
				y_offset += lmImgVal * ( *tmpYKernPtr++ );
			}
		}

		float x_val = float(x) + x_offset / total_mass;
		float y_val = float(y) + y_offset / total_mass;

		Particle p( x_val, y_val, total_mass, sliceIdx );
		return p;
	}
};

struct ParticleMatcher
{
	int N;
	int M;
	int sliceIdx;
	int maxStackCount;
	float neighborRadius;

	int * cellLowerBound;
	int * cellUpperBound;

	Particle* prevParticles;

	ParticleMatcher( int n, int m, int s, int mSC, int nR, int * cLB, int * cUB, Particle * pP ) :
		N( n ),
		M( m ),
		sliceIdx( s ),
		maxStackCount( mSC ),
		neighborRadius( nR ),
		cellLowerBound( cLB ),
		cellUpperBound( cUB ),
		prevParticles( pP )
	{
	}

	// Returns null if no match is found
	__host__ __device__
	Particle * operator()( Particle newParticle )
	{
		// There are a total of 9 cells we might have to search. last is sentinel
		int cellIndices[10]{ -1 };

		// But we always search at least one
		cellIndices[0] = pixelToGridIdx( newParticle, N, M );

		// Neighbors to follow
		Particle * pBestMatch = nullptr;
		for ( int c = 0; cellIndices[c] >= 0; c++ )
		{
			// It would be nice to parallelize around this, but probably not worth it
			int cellIdx = cellIndices[c];
			int lower = cellLowerBound[cellIdx];
			int upper = cellUpperBound[cellIdx];
			for ( int p = lower; p < upper; p++ )
			{
				Particle& oldParticle = prevParticles[p];

				// tooFar might not be necessary if I cull beforehand
				bool tooFar = ( sliceIdx - oldParticle.lastContributingsliceIdx != 1 );
				bool tooMany = ( oldParticle.nContributingParticles > maxStackCount );
				bool alreadyDone = ( oldParticle.pState == Particle::State::SEVER );
				if ( tooFar || tooMany || alreadyDone )
					continue;

				// See if the particle is within our range
				float dX = oldParticle.x - newParticle.x;
				float dY = oldParticle.y - newParticle.y;
				float distSq = pow( dX, 2 ) + pow( dY, 2 );

				if ( distSq < neighborRadius * neighborRadius )
				{
					// If there already was a match, see if this one is better
					if ( pBestMatch )
					{
						// Find the old distance
						dX = pBestMatch->x - newParticle.x;
						dY = pBestMatch->y - newParticle.y;

						// If this one is closer, assign it as the match
						if ( pow( dX, 2 ) + pow( dY, 2 ) > distSq )
							pBestMatch = &oldParticle;
					}
					else 
						pBestMatch = &oldParticle;
				}
			}
		}

		// Could check sever state here

		return pBestMatch;
	}
};

struct CheckIfMatchIsNotNull
{
	template <typename tuple_t>
	__host__ __device__
	bool operator()( const tuple_t T )
	{
		Particle * pMatch = thrust::get<1>( T );
		return pMatch != nullptr;
	}
};

// This gets called on matched particles and handles intensity state logic
// You should ensure this is thread safe beforehand, somehow (remove duplicates? not really sure)
struct UpdateMatchedParticle
{
	int sliceIdx;

	UpdateMatchedParticle( int s ) : sliceIdx( s ) {}

	// This kind of thing could be parallelized in a smarter way, probably
	template <typename tuple_t>
	__host__ __device__
	int operator()( const tuple_t T )
	{
		Particle newParticle = thrust::get<0>( T );
		Particle * pBestMatch = thrust::get<1>( T );
		switch ( pBestMatch->pState )
		{
			case Particle::State::NO_MATCH:
				// Shouldn't ever get no match, but assign the state and fall through
				pBestMatch->pState = Particle::State::INCREASING;
			case Particle::State::INCREASING:
				// If we're increasing, see if the new guy prompts a decrease
				// Should we check to see if more than one particle has contributed?
				if ( pBestMatch->i > newParticle.i )
					pBestMatch->pState = Particle::State::DECREASING;
				// Otherwise see if we should update the peak intensity and z position
				else if ( newParticle.i > pBestMatch->peakIntensity )
				{
					pBestMatch->peakIntensity = newParticle.i;
					pBestMatch->z = (float) sliceIdx;
				}
				break;
			case Particle::State::DECREASING:
				// In this case, if it's still decreasing then fall through
				if ( pBestMatch->i > newParticle.i )
					break;
				// If we're severing, assing the state and fall through
				pBestMatch->pState = Particle::State::SEVER;

				// I could probably catch this earlier
			case Particle::State::SEVER:
				// Continue here (could catch this earlier)
				pBestMatch = nullptr;
		}

		// could do this in yet another call, if you were so inclined
		// If we didn't sever and null out above
		if ( pBestMatch != nullptr )
		{
			// It's a match, bump the particle count and compute an averaged position (?)
			pBestMatch->nContributingParticles++;
			pBestMatch->lastContributingsliceIdx = sliceIdx;

			// I don't know about the averaged position thing
			pBestMatch->x = 0.5f * ( pBestMatch->x + newParticle.x );
			pBestMatch->y = 0.5f * ( pBestMatch->y + newParticle.y );
		}

		return 0;
	}
};

struct IsParticleUnmatched
{
	__host__ __device__
	bool operator()( const Particle p )
	{
		return p.pState == Particle::State::NO_MATCH;
	}
};

struct ParticleOrderingComp
{
	int N, M;
	ParticleOrderingComp( int n, int m ) : N( n ), M( m ) {}

	__host__ __device__
	bool operator()( const Particle a, const Particle b )
	{
		return pixelToGridIdx( a, N, M ) < pixelToGridIdx( b, N, M );
	}
};

struct MaybeRemoveParticle
{
	int sliceIdx;
	int minSlices;
	MaybeRemoveParticle( int s, int m ) : sliceIdx( s ), minSlices( m ) {}

	__host__ __device__
	bool operator()(const Particle p)
	{
		return ( sliceIdx - p.lastContributingsliceIdx > 2 && ( p.pState != Particle::State::SEVER || p.nContributingParticles < minSlices ) );
	}
};

uint32_t Solver::FindParticles( Datum& D )
{
	const int N = D.d_LocalMaxImg.rows;
	const int m = m_nMaxLevel;
	const int cellSize = N >> m;
	const int cellCount = N / cellSize;
	const int nTotalCells = cellCount * cellCount;

	// Make device pointers to the kernels used in particle solving and the localmax img
	using dFloatptr = thrust::device_ptr < float > ;
	dFloatptr d_pLocalMaxImgBuf( D.d_LocalMaxImg.ptr<float>() );
	dFloatptr d_pCirleKernel( m_dCircleMask.ptr<float>() );
	dFloatptr d_pRxKernel( m_dRadXKernel.ptr<float>() );
	dFloatptr d_pRyKernel( m_dRadYKernel.ptr<float>() );
	dFloatptr d_pR2Kernel( m_dRadSqKernel.ptr<float>() );

	// Cull the herd
	int minSlices = 3;
	auto itLastPrevParticle = thrust::remove_if( md_PrevParticleVec.begin(), md_PrevParticleVec.end(), MaybeRemoveParticle( D.sliceIdx, minSlices ) );

	// Make a device vector out of the particle buffer pointer (it's contiguous)
	thrust::device_ptr<unsigned char> d_pParticleImgBuf( D.d_ParticleImg.ptr<unsigned char>() );
	thrust::device_vector<unsigned char> d_ParticleImgVec( d_pParticleImgBuf, d_pParticleImgBuf + D.d_ParticleImg.size().area() );

	// For each pixel in the particle image, we care if it's nonzero and if it's far enough from the edges
	// So we need its index (transformable into twoD pos) and its value
	auto itDetectParticleBegin = thrust::make_zip_iterator( thrust::make_tuple( d_ParticleImgVec.begin(), thrust::counting_iterator<int>( 0 ) ) );
	auto itDetectParticleEnd = thrust::make_zip_iterator( thrust::make_tuple( d_ParticleImgVec.end(), thrust::counting_iterator<int>( N ) ) );

	// Do a stream compaction to get the nonzero particle locations
	// This vector is far too large, but I guess that's ok (you can keep it static in mem if you want)
	thrust::device_vector<Particle> d_NewParticleVec( N );
	auto itLastNewParticle = thrust::transform_if( itDetectParticleBegin, itDetectParticleEnd, d_NewParticleVec.begin(),
												   MakeParticleFromIdx( D.sliceIdx, N, m_uFeatureRadius, d_pLocalMaxImgBuf.get(), d_pCirleKernel.get(), d_pRxKernel.get(), d_pRyKernel.get(), d_pR2Kernel.get() ),
												   IsParticleAtIdx( N, m_uFeatureRadius ) );
	int newParticleCount = itLastNewParticle - d_NewParticleVec.begin();

	// The grid cell vec might be split into two vecs like this (they should also be class members, but I'll do that later)
	thrust::device_vector<int> d_GridCellLowerBoundsVec( nTotalCells ), d_GridCellUpperBoundsVec( nTotalCells );

	// Initialize grid cells
	using particleIter = thrust::device_vector<Particle>::iterator;
	using pixelToGridIdxIter = thrust::transform_iterator < PixelToGridIdx, particleIter > ;

	pixelToGridIdxIter itPrevParticleBegin = thrust::make_transform_iterator<PixelToGridIdx, particleIter>( md_PrevParticleVec.begin(), PixelToGridIdx( N, m ) );
	pixelToGridIdxIter itPrevParticleEnd = thrust::make_transform_iterator<PixelToGridIdx, particleIter>( itLastPrevParticle, PixelToGridIdx( N, m ) );

	thrust::lower_bound( itPrevParticleBegin, itPrevParticleEnd, thrust::counting_iterator<int>( 0 ), thrust::counting_iterator<int>( nTotalCells ), d_GridCellLowerBoundsVec.begin() );
	thrust::upper_bound( itPrevParticleBegin, itPrevParticleEnd, thrust::counting_iterator<int>( 0 ), thrust::counting_iterator<int>( nTotalCells ), d_GridCellUpperBoundsVec.begin() );
	

	// Tranform new particles into a vector of particle pointers; if they are null then no match was found (?)
	thrust::device_vector<Particle *> d_ParticleMatchVec( newParticleCount );
	// Note that I'm using itLastNewParticle
	thrust::transform( d_NewParticleVec.begin(), itLastNewParticle, d_ParticleMatchVec.begin(),
					   ParticleMatcher( N, m, D.sliceIdx, m_uMaxStackCount, m_uNeighborRadius, d_GridCellLowerBoundsVec.data().get(), d_GridCellUpperBoundsVec.data().get(), md_PrevParticleVec.data().get() ) );

	// Zip the pointer vec and newparticle vec
	auto itNewParticleToMatchedParticleBegin = thrust::make_zip_iterator( thrust::make_tuple( d_NewParticleVec.begin(), d_ParticleMatchVec.begin() ) );
	auto itNewParticleToMatchedParticleEnd = thrust::make_zip_iterator( thrust::make_tuple( itLastNewParticle, d_ParticleMatchVec.end() ) );

	// If there was a match, update the intensity state. I don't know how to do a for_each_if other than a transform_if that discards the output
	thrust::transform_if( itNewParticleToMatchedParticleBegin, itNewParticleToMatchedParticleEnd, thrust::discard_iterator<>(), UpdateMatchedParticle( D.sliceIdx ), CheckIfMatchIsNotNull() );

	// Tack the unmatched particles onto the endof the vector and sort the whole thing (do this with a merge). Reassign iterator to new end
	itLastPrevParticle = thrust::copy_if( d_NewParticleVec.begin(), d_NewParticleVec.end(), itLastPrevParticle, IsParticleUnmatched() );

	// Note that the above won't work, since the output vec must be resized. You need to get that count somewhere along the way, or oversize

	// Sort the new collection
	thrust::sort( md_PrevParticleVec.begin(), itLastPrevParticle, ParticleOrderingComp( N, m ) );

	return md_PrevParticleVec.size();
}

std::vector<Particle> Solver::GetFoundParticles() const
{

	//int nParticles = std::count_if( m_vPrevParticles.begin(), m_vPrevParticles.end(), [] ( const Particle& p ) {
	//	return p.pState == Particle::State::SEVER && p.nContributingParticles > 2;
	//} );
	//std::cout << "Final particle count: " << nParticles << std::endl;

	std::vector<Particle> ret;
	//( m_vFoundParticles.size() );
	//std::transform( m_vFoundParticles.begin(), m_vFoundParticles.end(), ret.begin(),
	//				[] ( const ParticleStack& pS ) {return pS.GetRefinedParticle(); } );
	return ret;
}