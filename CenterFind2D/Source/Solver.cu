#include "hip/hip_runtime.h"
#include "Solver.cuh"
#include "SolverOperators.cuh"

#include <iterator>
#include <algorithm>
#include <set>

#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>

#include <thrust/binary_search.h>
#include <thrust/sort.h>

#include <thrust/copy.h>
#include <thrust/transform.h>
#include <thrust/transform.h>

#include <thrust/functional.h>

#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/counting_iterator.h>

// x, y, i default to -1
__host__ __device__
Particle::Particle( float x, float y, float i, int idx ) :
z( idx ),
peakIntensity( i ),
nContributingParticles( 1 ),
lastContributingsliceIdx( idx ),
pState( Particle::State::NO_MATCH )
{
	this->x = x;
	this->y = y;
	this->i = i;
}

Solver::Solver() :
m_uMaskRadius( 0 ),
m_uFeatureRadius( 0 ),
m_uMinSliceCount( 0 ),
m_uMaxSliceCount( 0 ),
m_uNeighborRadius( 0 ),
m_uCurPrevParticleCount( 0 )
{
}

Solver::Solver( int mR, int fR, int minSC, int maxSC, int nR ) :
m_uMaskRadius( mR ),
m_uMaxLevel( 3 ),
m_uFeatureRadius( fR ),
m_uMinSliceCount( minSC ),
m_uMaxSliceCount( maxSC ),
m_uNeighborRadius( nR ),
m_uCurPrevParticleCount( 0 )
{
	// Neighbor region diameter
	int diameter = 2 * m_uMaskRadius + 1;

	// Make host mats
	cv::Mat h_Circ( cv::Size( diameter, diameter ), CV_32F, 0.f );
	cv::Mat h_RX( cv::Size( diameter, diameter ), CV_32F, 0.f );
	cv::Mat h_RY( cv::Size( diameter, diameter ), CV_32F, 0.f );
	cv::Mat h_R2( cv::Size( diameter, diameter ), CV_32F, 0.f );

	// set up circle mask
	cv::circle( h_Circ, cv::Point( m_uMaskRadius, m_uMaskRadius ), m_uMaskRadius, 1.f, -1 );

	// set up Rx and part of r2
	for ( int y = 0; y < diameter; y++ )
	{
		for ( int x = 0; x < diameter; x++ )
		{
			cv::Point p( x, y );
			h_RX.at<float>( p ) = x + 1;
			h_RY.at<float>( p ) = y + 1;
			h_R2.at<float>( p ) = pow( -(float) m_uMaskRadius + x, 2 ) + pow( -(float) m_uMaskRadius + y, 2 );
		}
	}

	// I forget what these do...
	cv::threshold( h_R2, h_R2, pow( (double) m_uMaskRadius, 2 ), 1, cv::THRESH_TOZERO_INV );
	cv::multiply( h_RX, h_Circ, h_RX );
	cv::multiply( h_RY, h_Circ, h_RY );

	/// For host debugging
	//h_Circ.copyTo( m_dCircleMask );
	//h_RX.copyTo( m_dRadXKernel );
	//h_RY.copyTo( m_dRadYKernel );
	//h_R2.copyTo( m_dRadSqKernel );

	// Upload to continuous gpu mats
	m_dCircleMask = getContinuousGpuMat( h_Circ );
	m_dRadXKernel = getContinuousGpuMat( h_RX );
	m_dRadYKernel = getContinuousGpuMat( h_RY );
	m_dRadSqKernel = getContinuousGpuMat( h_R2 );
}

template <typename ... Args>
auto makeZipIt( const Args&... args ) -> decltype( thrust::make_zip_iterator( thrust::make_tuple( args... ) ) )
{
	return thrust::make_zip_iterator( thrust::make_tuple( args... ) );
}

// This function removes particles from the vector of previously found particles if they 
// pass the predicate MaybeRemoveParticle
size_t Solver::cullExistingParticles( int curSliceIdx )
{
	size_t u_preremovePrevParticleCount = m_uCurPrevParticleCount;
	auto itLastPrevParticleEnd = m_dPrevParticleVec.begin() + m_uCurPrevParticleCount;
	auto itCurPrevParticleEnd = thrust::remove_if( m_dPrevParticleVec.begin(), itLastPrevParticleEnd, MaybeRemoveParticle( curSliceIdx, m_uMinSliceCount ) );
	m_uCurPrevParticleCount = itCurPrevParticleEnd - m_dPrevParticleVec.begin();
	size_t nRemovedParticles = u_preremovePrevParticleCount - m_uCurPrevParticleCount;

	return nRemovedParticles;
}

// Given the processed particle image, this function finds the particle locations and returns a vector of Particle objects
Solver::ParticleVec Solver::findNewParticles( UcharVec& d_ParticleImgVec, Floatptr pThreshImg, int N, int sliceIdx )
{
	// Create pointers to our kernels
	Floatptr d_pCirleKernel( (float *) m_dCircleMask.data );
	Floatptr d_pRxKernel( (float *) m_dRadXKernel.data );
	Floatptr d_pRyKernel( (float *) m_dRadYKernel.data );
	Floatptr d_pR2Kernel( (float *) m_dRadSqKernel.data );

	// For each pixel in the particle image, we care if it's nonzero and if it's far enough from the edges
	// So we need its index (transformable into twoD pos) and its value, which we zip
	auto itDetectParticleBegin = thrust::make_zip_iterator( thrust::make_tuple( d_ParticleImgVec.begin(), thrust::counting_iterator<int>( 0 ) ) );
	auto itDetectParticleEnd = thrust::make_zip_iterator( thrust::make_tuple( d_ParticleImgVec.end(), thrust::counting_iterator<int>( N*N ) ) );

	// Then, if the particle fits our criteria, we copy its index (from the counting iterator) into this vector, and discard the uchar
	IntVec d_NewParticleIndicesVec( N*N );
	auto itFirstNewParticle = thrust::make_zip_iterator( thrust::make_tuple( thrust::discard_iterator<>(), d_NewParticleIndicesVec.begin() ) );
	auto itLastNewParticle = thrust::copy_if( itDetectParticleBegin, itDetectParticleEnd, itFirstNewParticle, IsParticleAtIdx( N, m_uFeatureRadius ) );
	size_t newParticleCount = itLastNewParticle - itFirstNewParticle;

	// Now transform each index into a particle by looking at values inside the lmimg and using the kernels
	ParticleVec d_NewParticleVec( newParticleCount );
	thrust::transform( d_NewParticleIndicesVec.begin(), d_NewParticleIndicesVec.begin() + newParticleCount, d_NewParticleVec.begin(),
					   MakeParticleFromIdx( sliceIdx, N, m_uMaskRadius, pThreshImg.get(), d_pCirleKernel.get(), d_pRxKernel.get(), d_pRyKernel.get(), d_pR2Kernel.get() ) );

	return d_NewParticleVec;
}

// This function recreates the grid cell ranges given the current container of previous particles
void Solver::createGridCells( int N )
{
	// We don't bother if there are no previous particles
	if ( m_dPrevParticleVec.empty() )
		return;

	// If our grid cell vectors are empty, create them now
	if ( m_dGridCellLowerBoundVec.empty() || m_dGridCellUpperBoundVec.empty() )
	{
		const int cellSize = N >> m_uMaxLevel;
		const int cellCount = N / cellSize;
		const int nTotalCells = cellCount * cellCount;
		m_dGridCellLowerBoundVec.resize( nTotalCells );
		m_dGridCellUpperBoundVec.resize( nTotalCells );
	}

	// Some typedefs, we use a transform iterator to convert particles into indices
	using particleIter = ParticleVec::iterator;
	using pixelToGridIdxIter = thrust::transform_iterator < PixelToGridIdx, particleIter >;

	// Create an iterator to the end of our current previous particle container (might not be m_dPrevParticleVec.end())
	auto itCurPrevParticleEnd = m_dPrevParticleVec.begin() + m_uCurPrevParticleCount;

	// Create the transform iterator that iterates over our previous particles and returns their grid indices
	pixelToGridIdxIter itPrevParticleBegin = thrust::make_transform_iterator<PixelToGridIdx, particleIter>( m_dPrevParticleVec.begin(), PixelToGridIdx( N, m_uMaxLevel ) );
	pixelToGridIdxIter itPrevParticleEnd = thrust::make_transform_iterator<PixelToGridIdx, particleIter>( itCurPrevParticleEnd, PixelToGridIdx( N, m_uMaxLevel ) );

	// Find the ranges of previous particless
	const size_t nTotalCells = m_dGridCellLowerBoundVec.size();
	thrust::lower_bound( itPrevParticleBegin, itPrevParticleEnd, thrust::counting_iterator<int>( 0 ), thrust::counting_iterator<int>( nTotalCells ), m_dGridCellLowerBoundVec.begin() );
	thrust::upper_bound( itPrevParticleBegin, itPrevParticleEnd, thrust::counting_iterator<int>( 0 ), thrust::counting_iterator<int>( nTotalCells ), m_dGridCellUpperBoundVec.begin() );
}

// For each new particle, given the range of previous particles to search through, find the best match and return a pointer to its address
// If the pointer is null, then no match was found
Solver::ParticlePtrVec Solver::findParticleMatches( ParticleVec& d_NewParticleVec, int N, int sliceIdx )
{
	ParticlePtrVec d_ParticleMatchVec( d_NewParticleVec.size(), (Particle *)nullptr );

	// Only go through this is there are cells we could match with
	if ( m_dPrevParticleVec.empty() == false )
		thrust::transform( d_NewParticleVec.begin(), d_NewParticleVec.end(), d_ParticleMatchVec.begin(),
		ParticleMatcher( N, m_uMaxLevel, sliceIdx, m_uMaxSliceCount, m_dGridCellLowerBoundVec.size(), m_uNeighborRadius, m_dGridCellLowerBoundVec.data().get(), m_dGridCellUpperBoundVec.data().get(), m_dPrevParticleVec.data().get() ) );

	return d_ParticleMatchVec;
}

// For every matched particle, update its intensity state / position
void Solver::updateMatchedParticles( ParticleVec& d_NewParticleVec, ParticlePtrVec& d_ParticleMatchVec, int sliceIdx )
{
	// Zip the pointer vec and newparticle vec
	auto itNewParticleToMatchedParticleBegin = thrust::make_zip_iterator( thrust::make_tuple( d_NewParticleVec.begin(), d_ParticleMatchVec.begin() ) );
	auto itNewParticleToMatchedParticleEnd = thrust::make_zip_iterator( thrust::make_tuple( d_NewParticleVec.end(), d_ParticleMatchVec.end() ) );

	// If there was a match, update the intensity state. I don't know how to do a for_each_if other than a transform_if that discards the output
	thrust::transform_if( itNewParticleToMatchedParticleBegin, itNewParticleToMatchedParticleEnd, thrust::discard_iterator<>(), UpdateMatchedParticle( sliceIdx ), CheckIfMatchIsNotNull() );

#if _DEBUG
	// Useful for me to know how these start to spread out on debug
	//size_t numInNoMatch = thrust::count_if( m_dPrevParticleVec.begin(), m_dPrevParticleVec.end(), IsParticleInState<Particle::State::NO_MATCH>() );
	//size_t numInIncreasing = thrust::count_if( m_dPrevParticleVec.begin(), m_dPrevParticleVec.end(), IsParticleInState<Particle::State::INCREASING>() );
	//size_t numInDecreasing = thrust::count_if( m_dPrevParticleVec.begin(), m_dPrevParticleVec.end(), IsParticleInState<Particle::State::DECREASING>() );
	//size_t numInSever = thrust::count_if( m_dPrevParticleVec.begin(), m_dPrevParticleVec.end(), IsParticleInState<Particle::State::SEVER>() );
#endif
}

// For the particles that weren't matched, stream compact them into a vector and return it
Solver::ParticleVec Solver::consolidateUnmatchedParticles( ParticleVec& d_NewParticleVec, ParticlePtrVec& d_ParticleMatchVec )
{
	// Zip the pointer vec and newparticle vec
	auto itNewParticleToMatchedParticleBegin = thrust::make_zip_iterator( thrust::make_tuple( d_NewParticleVec.begin(), d_ParticleMatchVec.begin() ) );
	auto itNewParticleToMatchedParticleEnd = thrust::make_zip_iterator( thrust::make_tuple( d_NewParticleVec.end(), d_ParticleMatchVec.end() ) );

	// Copy all unmatched particles into a new vector; we copy a tuple of new particles and pointers to matches, discarding the pointers
	ParticleVec d_UnmatchedParticleVec( d_NewParticleVec.size() );
	auto itNewParticleAndPrevParticleMatchBegin = thrust::make_zip_iterator( thrust::make_tuple( d_UnmatchedParticleVec.begin(), thrust::discard_iterator<>() ) );

	// Copy new particles if their corresponding match is null
	auto itNewParticleAndPrevParticleMatchEnd = thrust::copy_if( itNewParticleToMatchedParticleBegin, itNewParticleToMatchedParticleEnd, itNewParticleAndPrevParticleMatchBegin, CheckIfMatchIsNull() );
	size_t numUnmatchedParticles = itNewParticleAndPrevParticleMatchEnd - itNewParticleAndPrevParticleMatchBegin;

	// Size down and return
	d_UnmatchedParticleVec.resize( numUnmatchedParticles );
	return d_UnmatchedParticleVec;
}

// Given our previous particles and the newly found unmatched particles, merge them into a sorted container
void Solver::mergeUnmatchedParticles( ParticleVec& d_UnmatchedParticleVec, int N )
{
	// We have two options here; the easy option is to just tack these new particles onto the previous particle vector and sort the whole thing
	// alternatively you could set a flag in previous particles if the matching process caused them to move to a new grid cell and then treat those particles as unmatched
	// you could then sort the unmatched particles (relatively few compared to the count of previous particles) and then merge them into the prev particle vec, which is still sorted
	// Below is the first option, whihc was easier.

	// first make room for the new particles, if we need it
	size_t newPrevParticleCount = d_UnmatchedParticleVec.size() + m_uCurPrevParticleCount;
	if ( newPrevParticleCount > m_dPrevParticleVec.size() )
		m_dPrevParticleVec.resize( newPrevParticleCount );

	// copy unmatched particles onto the original end of the previous particle vec
	auto itNewParticleDest = m_dPrevParticleVec.begin() + m_uCurPrevParticleCount;
	auto itEndOfPrevParticles = thrust::copy( d_UnmatchedParticleVec.begin(), d_UnmatchedParticleVec.end(), itNewParticleDest );

	// Sort the whole thing
	thrust::sort( m_dPrevParticleVec.begin(), itEndOfPrevParticles, ParticleOrderingComp( N, m_uMaxLevel ) );
	m_uCurPrevParticleCount = newPrevParticleCount;
}

int Solver::FindParticles( Datum& D )
{
	// We assume the row and column dimensions are equal
	const int N = D.d_InputImg.rows;

	// Make a device vector out of the particle buffer pointer (it's contiguous)
	UcharPtr d_pParticleImgBufStart( (unsigned char *) D.d_ParticleImg.datastart );
	UcharPtr d_pParticleImgBufEnd( (unsigned char *) D.d_ParticleImg.dataend );
	UcharVec d_ParticleImgVec( d_pParticleImgBufStart, d_pParticleImgBufEnd );
	
	/// For host debugging
	//cv::Mat h_ThreshImg;
	//D.d_ThreshImg.download( h_ThreshImg );
	//Floatptr d_pLocalMaxImgBuf( h_ThreshImg.ptr<float>() );
	Floatptr d_pThreshImgBuf( (float *) D.d_ThreshImg.data );

	Floatptr d_pCirleKernel( (float *) m_dCircleMask.data );
	Floatptr d_pRxKernel( (float *) m_dRadXKernel.data );
	Floatptr d_pRyKernel( (float *) m_dRadYKernel.data );
	Floatptr d_pR2Kernel( (float *) m_dRadSqKernel.data );

	// Cull the herd
	size_t numParticlesRemoved = cullExistingParticles( D.sliceIdx );

	// Find new particles
	ParticleVec d_NewParticleVec = findNewParticles( d_ParticleImgVec, d_pThreshImgBuf, N, D.sliceIdx );
	size_t numParticlesFound = d_NewParticleVec.size();

	// Initialize grid cells given current container of previous particles
	createGridCells( N );

	// Tranform new particles into a vector of particle pointers; if they are null then no match was found (?)
	ParticlePtrVec d_ParticleMatchVec = findParticleMatches( d_NewParticleVec, N, D.sliceIdx );

	// For particles we were able to match, update their intensity states
	updateMatchedParticles( d_NewParticleVec, d_ParticleMatchVec, D.sliceIdx );

	// Copy all unmatched particles into a new vector; we copy a tuple of new particles and pointers to matches, discarding the pointers
	ParticleVec d_UnmatchedParticleVec = consolidateUnmatchedParticles( d_NewParticleVec, d_ParticleMatchVec );

	// Merge unmatched particles into our container, preserving grid index order
	mergeUnmatchedParticles( d_UnmatchedParticleVec, N );

	std::cout << "Slice Idx:\t" << D.sliceIdx << "\tNew Particles:\t" << numParticlesFound << "\tUnmatched Particles:\t" << d_UnmatchedParticleVec.size() << "\tFound Particles:\t" << m_uCurPrevParticleCount << "\tCulled Particles:\t" << numParticlesRemoved << std::endl;

	return m_uCurPrevParticleCount;
}

std::vector<Particle> Solver::GetFoundParticles() const
{

	//int nParticles = std::count_if( m_vPrevParticles.begin(), m_vPrevParticles.end(), [] ( const Particle& p ) {
	//	return p.pState == Particle::State::SEVER && p.nContributingParticles > 2;
	//} );
	//std::cout << "Final particle count: " << nParticles << std::endl;

	std::vector<Particle> ret;
	//( m_vFoundParticles.size() );
	//std::transform( m_vFoundParticles.begin(), m_vFoundParticles.end(), ret.begin(),
	//				[] ( const ParticleStack& pS ) {return pS.GetRefinedParticle(); } );
	return ret;
}