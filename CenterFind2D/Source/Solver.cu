#include "hip/hip_runtime.h"
#include "CenterFind.h"

#include <set>

#include <hip/hip_runtime.h>

#include "ThrustOps.cuh"

// We need to change this so that most of the solving mechanisms are accessible via CUDA

Solver::Solver() :
m_uMaskRadius( 0 ),
m_uFeatureRadius( 0 ),
m_uMaxStackCount( 0 ),
m_uNeighborRadius( 0 )
{
}


struct get_int2 : public thrust::unary_function<unsigned int, int2 >
{
	int N;
	get_int2( int n ) :
		N( n )
	{
	}
	__host__ __device__
		inline int2 operator()( unsigned int idx )
	{
		int x = idx % N;
		int y = idx / N;;
		return make_int2( x, y );
	}
};

// Kernel for initializing solver kernels...kernel kernel
__global__
void createSolverKernels( int radius, float * circ, float * x, float * y, float * sq )
{
	// Pretty small sizes here
	int idx_X = threadIdx.x + blockDim.x * blockIdx.x;
	int idx_Y = threadIdx.y + blockDim.y * blockIdx.y;
	int diameter = 2 * radius + 1;
	int idx = idx_X + idx_Y * diameter;

	x[idx] = idx_X + 1;
	y[idx] = idx_Y + 1;
	sq[idx] = powf( x[idx] - idx_X, 2 ) + powf( y[idx] - idx_Y, 2 );

}

Solver::Solver( uint32_t mR, uint32_t fR, uint32_t minSC, uint32_t maxSC, uint32_t nR ) :
m_uMaskRadius( mR ),
m_uFeatureRadius( fR ),
m_uMinStackCount( minSC ),
m_uMaxStackCount( maxSC ),
m_uNeighborRadius( nR )
{
	// Neighbor region diameter
	int diameter = 2 * m_uMaskRadius + 1;

	// Create GpuMats and initialize via kernel
	auto makeContinuousGmat = [diameter] () {
		GpuMat g = cv::cuda::createContinuous( cv::Size( diameter, diameter ), CV_32F );
		assert( g.isContinuous() && "We need contiguous arrays here" );
		return g;
	};

	m_CircleMask = makeContinuousGmat();
	m_RadXKernel = makeContinuousGmat();
	m_RadYKernel = makeContinuousGmat();
	m_RadSqKernel = makeContinuousGmat();

	// make data
	dim3 gridSize( 1 ), blockSize( diameter, diameter );
	createSolverKernels << < gridSize, blockSize >> >( (int) m_uMaskRadius, m_CircleMask.ptr<float>(), m_RadSqKernel.ptr<float>(), m_RadYKernel.ptr<float>(), m_RadSqKernel.ptr<float>() );

	// Make host mats
	cv::Mat h_Circ( cv::Size( diameter, diameter ), CV_32F, 0.f );
	cv::Mat h_RX = h_Circ;
	cv::Mat h_RY = h_Circ;
	cv::Mat h_R2 = h_Circ;

	// set up circle mask
	cv::circle( h_Circ, cv::Point( m_uMaskRadius, m_uMaskRadius ), m_uMaskRadius, 1.f, -1 );

	// set up Rx and part of r2
	for ( int i = 0; i < diameter; i++ )
	{
		for ( int j = 0; j < diameter; j++ )
		{
			h_RX.at<float>( i, j ) = float( j + 1 );
			h_R2.at<float>( i, j ) += float( pow( j - m_uMaskRadius, 2 ) );
		}
	}

	// set up Ry and the rest of r2
	for ( int i = 0; i < diameter; i++ )
	{
		for ( int j = 0; j < diameter; j++ )
		{
			h_RY.at<float>( i, j ) = float( i + 1 );
			h_R2.at<float>( i, j ) += float( pow( i - m_uMaskRadius, 2 ) );
		}
	}

	// I forget what these do...
	cv::threshold( h_R2, h_R2, pow( (double) m_uMaskRadius, 2 ), 1, cv::THRESH_TOZERO_INV );
	cv::multiply( h_RX, h_Circ, h_RX );
	cv::multiply( h_RY, h_Circ, h_RY );

	auto helper = [] ( cv::Mat& m ) {
		GpuMat g = cv::cuda::createContinuous( m.size(), m.type() );
		if ( g.isContinuous() == false )
		{
			// ruh roh
		}
		// copy memory
		return g;
	};

	// Create contiguous GPU Mats for these

	m_CircleMask = cv::cuda::createContinuous( h_Circ.size(), h_Circ.type() );

	// copy these to contiguous GpuMats
	m_CircleMask.upload( h_Circ );
	m_RadXKernel.upload( h_RX );
	m_RadYKernel.upload( h_RY );
	m_RadSqKernel.upload( h_R2 );

}

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/random.h>
#include <thrust/copy.h>
#include <thrust/tabulate.h>
#include <thrust/sort.h>
#include <thrust/binary_search.h>

struct is_nonzero : public thrust::unary_function<thrust::tuple<unsigned char, int2>, bool >
{
	__host__ __device__
		inline bool operator()( const thrust::tuple<unsigned char, int2>& t )
	{
		return t.get<0>() != 0;
	}
};

__host__ __device__
int pLoc2Zcode( float2 pLoc, int max_level, int N )
{
	float xMin = 0, yMin = 0, xMax = N, yMax = N;

	int result = 0;

	for ( int level = 1; level <= max_level; level++ )
	{
		// Classify in x-direction
		float xmid = 0.5f * ( xMin + xMax );
		int x_hi_half = ( p.x < xmid ) ? 0 : 1;

		// Push the bit into the result as we build it
		result |= x_hi_half;
		result <<= 1;

		// Classify in y-direction
		float ymid = 0.5f * ( yMin + yMax );
		int y_hi_half = ( p.y < ymid ) ? 0 : 1;

		// Push the bit into the result as we build it
		result |= y_hi_half;
		result <<= 1;

		// Shrink the bounding box, still encapsulating the point
		xMin = ( x_hi_half ) ? xmid : xMin;
		xMax = ( x_hi_half ) ? xMax : xmid;
		yMin = ( y_hi_half ) ? ymid : yMin;
		yMax = ( y_hi_half ) ? yMax : ymid;
	}

	result >>= 1;
	return result;
}

struct Particle
{
	float2 pos;
	float intensity;
	int zCode;

	__host__ __device__
	Particle( float2 p, float i, int z ) :
		pos( p ),
		intensity( i ),
		zCode( z )
	{
	}
};

struct ParticleComp
{
	__host__ __device__
	bool operator()( Particle a, Particle b )
	{
		return a.zCode < b.zCode;
	}
};

// This is the second filter; once we know a particle is non-zero, we have to do a local sum around it
// to determine its "mass" (or intensity, not really sure)
struct GetParticle
{
	// Kernel Radius
	int kernelRadius;

	// Image dimensions
	int N;

	// offset kernels
	float * circKernel;
	float * xKernel;
	float * yKernel;
	float * sqKernel;

	// The actual reference images we multiply against
	float * lmImg;

	__host__ __device__
	GetParticle( int kD, int N, float * lmImg, float * cK, float * xK, float * yK, float * sqK ) :
		kernelRadius( kD ),
		N( N ),
		lmImg( lmImg ),
		circKernel( cK ),
		xKernel( xK ),
		yKernel( yK ),
		sqKernel( sqK )
	{
	}
	

	__host__ __device__
	Particle operator()( int idx )
	{
		// This would be the 2-d pixel location
		int2 loc2D = get_int2( N )( idx );

		// Center of the sum region
		float * center = &lmImg[idx];
		float total_mass( 0 );
		float x_Offset( 0 ), y_Offset( 0 ), sq_Offset( 0 );

		// I need to do the arithmetic that lets me loop through the square around center

		// Get the total mass and unnormalized x,y,sq offsets
		for ( int i = 0; i < 2 * kernelRadius + 1; i++ )
		{
			total_mass += circKernel[i] * center[i];
			x_Offset += xKernel[i] * center[i];
			y_Offset += yKernel[i] * center[i];
			sq_Offset += sqKernel[i] * center[i];
		}

		x_Offset /= total_mass;
		y_Offset /= total_mass;
		sq_Offset /= total_mass;

		// Compute x,y positions
		float xVal = x_Offset + loc2D.x;
		float yVal = x_Offset + loc2D.y;
		float r2_val = sq_Offset;

		// particle location and z code
		float2 pLoc = make_float2( xVal, yVal );
		int zCode = pLoc2Zcode( pLoc, 3, N );

		// Construct and return particle
		Particle p( pLoc, total_mass, zCode );
		return p;
	}
};

uint32_t Solver::FindParticles( Datum& D )
{
	// The particle image is contiguous, so let's find all particle locations and store their index in the image
	int N = D.d_ParticleImg.size().area();
	thrust::device_vector<int> d_ParticleIndices( N );

	// First make a device vector out of the existing particle image
	thrust::device_ptr<unsigned char> d_ParticleImgPtr( D.d_ParticleImg.ptr() );
	thrust::device_vector<unsigned char> d_ParticleImgVec( d_ParticleImgPtr, d_ParticleImgPtr + N );

	// Now we must zip the iterators such that every time we find a non-zero particle pixel, we're also given its location
	auto locFindItBegin = thrust::make_zip_iterator( thrust::make_tuple( d_ParticleImgVec.begin(), thrust::counting_iterator<int>(0) ) );
	auto locFindItEnd = thrust::make_zip_iterator( thrust::make_tuple( d_ParticleImgVec.end(), thrust::counting_iterator<int>( N ) ) );
	
	// The output iterator throws away the unsigned char img pixel values using a discard iterator, so we're left with the int2s
	auto locFindItOutput = thrust::make_zip_iterator( thrust::make_tuple( thrust::discard_iterator<>(), d_ParticleIndices.begin() ) );

	// Stream compact locations
	auto lastParticleIt = thrust::copy_if( locFindItBegin, locFindItEnd, locFindItOutput, is_nonzero() ); // is this legit? if not is_nonzero() works
	size_t numParticles = lastParticleIt - locFindItOutput;

	// Also, if you ever decide to display the particle locations on an image, here are the 2-d locations
	thrust::device_vector<int2> d_2DParticleLocations( numParticles );
	thrust::transform( d_ParticleIndices.begin(), d_ParticleIndices.end(), d_2DParticleLocations.begin(), get_int2( sqrt( N + 0.1 ) ) );

	// For each newly found particle, we can now transform it into a real particle
	// In order to do that we'll need some info about the lm img
	float * lmImg = D.d_LocalMaxImg.ptr<float>();
	float * circKern = m_CircleMask.ptr<float>();
	float * xKern = m_RadXKernel.ptr<float>();
	float * yKern = m_RadYKernel.ptr<float>();
	float * sqKern = m_RadSqKernel.ptr<float>();
	GetParticle gPOp( (int)m_uMaskRadius, (int)sqrt( N + 0.1 ), lmImg, circKern, xKern, yKern, sqKern );
	thrust::device_vector<Particle> d_ParticleVec( numParticles );
	thrust::transform( d_ParticleIndices.begin(), d_ParticleIndices.end(), d_ParticleVec.begin(), gPOp );

	// This is a dummy vector that would contain all previously found particles (a work in progress), sorted by their z-code
	thrust::device_vector<Particle> d_PreviouslyFoundParticleVec;

	// Find the range of previously found particles that could match our newly found particles
	thrust::device_vector<int> d_PrevParticleLB( numParticles ), d_PrevParticleUB( numParticles );
	thrust::lower_bound( d_PreviouslyFoundParticleVec.begin(), d_PreviouslyFoundParticleVec.end(), d_PreviouslyFoundParticleVec.begin(), d_PreviouslyFoundParticleVec.end(), ParticleComp() );
	thrust::upper_bound( d_PreviouslyFoundParticleVec.begin(), d_PreviouslyFoundParticleVec.end(), d_PreviouslyFoundParticleVec.begin(), d_PreviouslyFoundParticleVec.end(), ParticleComp() );
}