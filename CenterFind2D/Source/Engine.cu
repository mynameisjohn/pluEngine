#include "hip/hip_runtime.h"
#include "CenterFind.h"
#include "FnPtrHelper.h"
#include "Solver.cuh"

#include <opencv2/cudaarithm.hpp>

#include <map>

#include <FreeImage.h>

Engine::Engine() {
}

// Must be implemented here to know what a Solver is (and destroy it)
Engine::~Engine()
{

}

int Engine::Execute() {
	// Return if no data
	if (m_vData.empty())
		return -1;

	// This lets the user set DSP params
	getUserParams(m_vData.front(), &m_fnBandPass, &m_fnLocalMax );

	m_ParticleSolver = std::unique_ptr<Solver>( new Solver( 3, m_fnBandPass.GetGaussianRadius(), 3, 5, 8 ) );

	// Run Centerfind algorithm on remaining images
	for (auto& D : m_vData) {
		m_fnBandPass(D);
		m_fnLocalMax(D);
		m_ParticleSolver->FindParticles(D);
	}

	//auto shit = m_ParticleSolver->GetFoundParticles();
	return 0;
}


void RemapImage(GpuMat& img, float m, float M) {
	float range = M - m;
	double min(1), max(2);
	cv::cuda::minMax(img, &min, &max);
	double alpha = range / (max - min);
	double beta = range * min / (max - min);
	double scale = range / (max - min);
	img.convertTo(img, CV_32F, alpha, beta);
}

void DisplayImage(GpuMat& img) {
	GpuMat disp;
	img.convertTo(disp, CV_32F);
	RemapImage(disp, 0, 1);
	std::string winName("disp");
	cv::namedWindow(winName, cv::WINDOW_OPENGL);
	cv::imshow(winName, disp);
	cv::waitKey();
}